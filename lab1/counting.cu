#include "hip/hip_runtime.h"
#include "counting.h"
#include "SyncedMemory.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <cstdlib>
#include <iostream>
#include <vector>

using namespace std;

__device__ __host__ int CeilDiv(int a, int b) {

	return (a - 1) / b + 1;
}
__device__ __host__ int CeilAlign(int a, int b) {

	return CeilDiv(a, b) * b;
}
__global__ void BuildTree_layer(const char* text, int* bottom, int bottom_size){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < bottom_size){
		if (text[idx] == '\n') bottom[idx] = 0;
		else bottom[idx] = 1;
		
		


	}

}
/*__global__ void BuildTree_other(const char* text, int *pos, int layersize){

int idx = blockIdx.x * blockDim.x + threadIdx.x;



}
*/

void CountPosition(const char *text, int *pos, int text_size)
{
	int bottom_size = text_size;
	vector<int*> perlayerStart; // record layer begin position.
	int TREE_index = 0;
	int *temp = 0;

	hipMalloc(&temp, sizeof(int)*bottom_size);
	perlayerStart.push_back(temp);
	int grid_dim = bottom_size >> 5 + 1;
	if (TREE_index == 0) BuildTree_layer << <grid_dim, 32 >> >(text, temp, bottom_size);










}

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead = 0;
	hipMalloc(&buffer, sizeof(int)*text_size * 2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer + text_size);

	// TODO

	hipFree(buffer);
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{







}
