#include "hip/hip_runtime.h"
#include "counting.h"
#include "SyncedMemory.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <cstdlib>
#include <iostream>
#include <vector>

using namespace std;

__device__ __host__ int CeilDiv(int a, int b) {

	return (a - 1) / b + 1;
}
__device__ __host__ int CeilAlign(int a, int b) {

	return CeilDiv(a, b) * b;
}
__global__ void BuildTree_1st(const char* text, int* bottom, int bottom_size){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < bottom_size){

		if (text[idx] == '\n') bottom[idx] = 0;
		else bottom[idx] = 1;
		//printf("%d, %d\n", idx, bottom[idx]);
	}

}
struct ifone{
	__host__ __device__ bool operator()(const int x){
		return (x == 1);
	}
};
__global__ void dropvector(char* text, int* pos, int text_size){
	
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	//printf("[i:%d=%c]",idx, text[idx]);

	char a = 0;
	if (idx % 2 == 0 && text[idx]!='\n' && text[idx+1]!='\n'){
		a = text[idx];
		text[idx] = text[idx + 1];
		text[idx + 1] = a;
	}



}
/*
__global__ int trace(){

	//check���S����
	//if ( idx /2 >= text_size/(hight+1)*2)
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int height = 0;
	int length = 0;


	//bottom up

	if (idx % 2 == 1){ //right node
		if (idx / 2 >= text_size / (hight + 1) * 2){
			while (idx /= 2, height += 1){
				if ([height][idx] != 0){ //dad not 0.
					if ((idx) % 2 == 0){// is dad right or left?
						//yes . dad is left.
						if ([height][idx - 1] == 0){ //is dad's left = 0?
							//yes. 
							length += [height][idx];
							idx = idx - 1;

							while ((idx * 2) + 1, height -= 1) {// top down
								if ([height][idx] != 0){
									length += [height][idx];
									return length;
								}
							}
						}
						else{//���������䤣�O�s
							continue;
						}
					}
					else{//dad is right
						length += [height][idx];
						return length;
					}
				}
				else{//dad is 0.
					length = 1;
					return length;
				}
				if (!(idx / 2 >= text_size / (hight + 1) * 2)) { //if no parent �ǩǪ�
					return length;
				}
			}
		}
		else{//���I�S����



		}
	}
	else{  //left node

		//���`�I ������
		//���� is 0?
		if ([idx - 1] != 0){ //�����D�s
			length += [height][idx];
		}
		else{//�������s
			length = 1;
			return length;

		}
		idx = idx - 1; //����
		if ((idx / 2) >= text_size / (hight + 1) * 2){ //if node HAVE parent
			while (idx /= 2, height += 1){
				if ([height][idx] != 0){ //dad not 0.
					if (idx % 2 == 0){// is dad right or left?
						//yes . dad is left.
						if ([height][idx - 1] == 0){ //is dad's left = 0?
							//yes. 
							//topdown
							length += [height][idx];
							while ((idx * 2) + 1, height -= 1) {
								if ([height][idx] != 0){
									length += [height][idx];
									return length;
								}
							}

						}
						else{
							length += [height][idx];
							return;
						}
					}

				}
				else {//if dad is 0.
					length[]
				}
				if (!(idx / 2 >= text_size / (hight + 1) * 2)) return;
			}
		}
		else{ // node NOT have parent
			return;
		}
	}

	else //no parent.
	{
		return;
	}


}*/

__global__ void BuildTree_other(int *Pre_layer, int *layer, int layertext){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < layertext){
		if (Pre_layer[2 * idx] & Pre_layer[2 * idx + 1])
			layer[idx] = Pre_layer[2 * idx + 1] + Pre_layer[2 * idx];
		else
			layer[idx] = 0;
		//printf("i=%d idx = %d, %d\n", layertext, idx, layer[idx]);
	}
}
void CountPosition(const char *text, int *pos, int text_size)
{
	//thrust::host_vector <int> H(4);
	int arrSize = text_size;
	int  *treeLayerBegin[9]; //record layer begin position.
	int** treeArray = 0;
	treeArray = (int **)malloc(9 * sizeof(int*));
	for (int i = 0; i < 9; i++){
		treeArray[i] = (int *)malloc(sizeof(int)*arrSize);
		treeLayerBegin[i] = treeArray[i];
		hipMalloc((void**)&treeLayerBegin[i], sizeof(int)*arrSize);
		arrSize = (arrSize % 2) ? (arrSize - 1) / 2 : arrSize / 2;
	}

	int blockdim = (text_size / 512) + 1; //16�n��512
	BuildTree_1st << <blockdim, 512 >> >(text, treeLayerBegin[0], text_size);

	int layertext = (text_size % 2) ? (text_size - 1) / 2 : text_size / 2;
	int a = 256;//??
	for (int i = 1; i < 9; i++){
		blockdim = (layertext) / 256 + 1;
		BuildTree_other << <blockdim, a >> > (treeLayerBegin[i - 1], treeLayerBegin[i], layertext);
		a /= 2;
		layertext /= 2;
	}



}




int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead = 0;
	hipMalloc(&buffer, sizeof(int)*text_size * 2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer + text_size);

	// TODO
	
	thrust::sequence(flag_d, cumsum_d);
	nhead = thrust::copy_if(flag_d,cumsum_d, pos_d, head_d,ifone()) - head_d;
	

	hipFree(buffer);
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{

	int blockdim = (text_size / 512) + 1; //16�n��512
	dropvector << <blockdim, 512 >> >(text, pos, text_size);





}
