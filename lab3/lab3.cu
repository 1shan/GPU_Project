#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>


__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int xt = blockIdx.x * blockDim.x + threadIdx.x; //target.x
	const int yt = blockIdx.y * blockDim.y + threadIdx.y; //target.y

	const int curt = wt*yt+xt; 
	//�ۤv�btarget����m
	//���W�Ltarget �B�bmask��mgray��>127.f
	if (yt < ht && xt < wt && mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;//�ۤv�bbackground����m
		if (0 <= yb && yb < hb && 0 <= xb && xb < wb) {
			output[curb * 3 + 0] = target[curt * 3 + 0];
			output[curb * 3 + 1] = target[curt * 3 + 1];
			output[curb * 3 + 2] = target[curt * 3 + 2];
		}
	}
}

__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *fixed,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	const int xt = blockDim.x * blockIdx.x + threadIdx.x;
	const int yt = blockDim.y * blockIdx.y + threadIdx.y;
	const int curt = wt * yt + xt;

	float t_sum, b_sum;

	if (yt < ht && xt < wt) {
		const int yb = oy + yt, xb = ox + xt;//background.x background.y
		const int curb = wb*yb + xb;//background.cu0ent

		for (int i = 0; i < 3; i++){
			t_sum = 0;
			b_sum = 0;

			//�ˬd�F�~�O�_�btarget��    

			if (xt - 1 >= 0){
				b_sum += mask[(curt - 1)] < 127.0 ? background[(curb - 1) * 3 + i] : 0;
				t_sum += target[(curt - 1) * 3 + i];
			}
			else{
				b_sum += background[(curb - 1) * 3 + i];
				t_sum += target[(curt)* 3 + i];
			}

			if (xt + 1 < wt){
				b_sum += mask[(curt + 1)] < 127.0 ? background[(curb + 1) * 3 + i] : 0;
				t_sum += target[(curt + 1) * 3 + i];
			}
			else{
				b_sum += background[(curb + 1) * 3 + i];
				t_sum += target[(curt)* 3 + i];
			}

			if (yt - 1 >= 0){
				b_sum += mask[(curt - wt)] < 127.0 ? background[(curb - wb) * 3 + i] : 0;
				t_sum += target[(curt - wt) * 3 + i];
			}
			else{
				b_sum += background[(curb - wb) * 3 + i];
				t_sum += target[(curt)* 3 + i];
			}

			if (yt + 1 < ht){
				b_sum += mask[(curt + wt)] < 127.0 ? background[(curb + wb) * 3 + i] : 0;
				t_sum += target[(curt + wt) * 3 + i];
			}
			else{
				b_sum += background[(curb + wb) * 3 + i];
				t_sum += target[(curt)* 3 + i];
			}

			fixed[curt * 3 + i] = 4.0*target[curt * 3 + i] - t_sum + b_sum;
		}
	}
}

__global__ void PoissonImageCloningInteration(
	const float *fixed,
	const float *mask,
	const float *target, //buf1
	float *output,		 //buf2
	const int wt,
	const int ht)
{

	const int xt = blockIdx.x * blockDim.x + threadIdx.x; //target.x
	const int yt = blockIdx.y * blockDim.y + threadIdx.y; //target.y
	const int curt = wt * yt + xt;
	float w = 1.4; //SOR parameters

	float neibor_sum = 0;
	if (yt < ht && xt < wt && mask[curt] > 127.0f){
		for (int i = 0; i < 3; i++){
			neibor_sum = 0;
			//���k�F�~���ժ����n++

			// West
			if (xt - 1 >= 0 && mask[(curt - 1)] > 127.0f){
				neibor_sum += target[(curt - 1) * 3 + i];
			}
			// East
			if (xt + 1 < wt && mask[(curt + 1)] > 127.0f){
				neibor_sum += target[(curt + 1) * 3 + i];
			}
			// North
			if (yt - 1 >= 0 && mask[(curt - wt)] > 127.0f){
				neibor_sum += target[(curt - wt) * 3 + i];
			}
			// South
			if (yt + 1 < ht && mask[(curt + wt)] > 127.0f){
				neibor_sum += target[(curt + wt) * 3 + i];
			}
			//* This part for original jacobi method * //
			//output[curt * 3 + i] = (fixed[curt * 3 + i] + neibor_sum) / 4;

			//* This part for  acceleration *//
			output[curt * 3 + i] = w * (fixed[curt * 3 + i] + neibor_sum) / 4 + (1.0 - w) * output[curt * 3 + i];
		}
	}
}
void PoissonImageCloning(
	const float *background, //Wb*Hb
	const float *target, //Wt*Ht
	const float *mask,// 1 channel, 0.0f/255.0f -> false/true. 
	float *output, //Wb*Hb
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	//hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	//SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
	//	background, target, mask, output,
	//	wb, hb, wt, ht, oy, ox
	//);

	//set up
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3 * wt*ht*sizeof(float));
	hipMalloc(&buf1 , 3 * wt*ht*sizeof(float));
	hipMalloc(&buf2 , 3 * wt*ht*sizeof(float));

	//initialize the iteration
	dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);
	CalculateFixed <<<gdim, bdim >>>(background, target, mask, fixed,wb, hb, wt, ht, oy, ox);

	hipMemcpy(buf1, target, sizeof(float) * 3 * wt * ht, hipMemcpyDeviceToDevice);

	//iterate
	for (int i = 0; i < 5000; ++i){
		PoissonImageCloningInteration <<<gdim, bdim >>>(fixed, mask, buf1, buf2, wt, ht);
		PoissonImageCloningInteration <<<gdim, bdim >>>(fixed, mask, buf2, buf1, wt, ht);
	}
	////copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float) * 3, hipMemcpyDeviceToDevice);
	SimpleClone <<<gdim, bdim >>>(background, buf1, mask, output, wb, hb, wt, ht, oy, ox);
	
	//clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
	
}































